#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>
#include "cmdline.h"
#include "input.h"
#include "config.h"
#include "timer.h"
#include "formats.h"
#include <hip/hip_runtime.h>


#define max(a,b) \
({ __typeof__ (a) _a = (a); \
   __typeof__ (b) _b = (b); \
 _a > _b ? _a : _b; })

#define min(a,b) \
({ __typeof__ (a) _a = (a); \
   __typeof__ (b) _b = (b); \
 _a < _b ? _a : _b; })
 
void usage(int argc, char** argv)
{
    printf("Usage: %s [my_matrix.mtx]\n", argv[0]);
    printf("Note: my_matrix.mtx must be real-valued sparse matrix in the MatrixMarket file format.\n"); 
}

void convert_to_csr(graph *g, csr_graph *csr) {
    csr->num_vertices = g->num_vertices;
    csr->row_offsets = (int *)malloc((g->num_vertices + 1) * sizeof(int));
    
    int edge_count = 0;
    for(int i = 0; i < g->num_vertices; ++i) {
        csr->row_offsets[i] = edge_count;
        node *temp = g->adj_lists[i];
        while(temp) {
            edge_count++;
            temp = temp->next;
        }
    }
    csr->row_offsets[g->num_vertices] = edge_count;
    
    csr->col_indices = (int *)malloc(edge_count * sizeof(int));
    csr->num_edges = edge_count;
    
    int idx = 0;
    for(int i = 0; i < g->num_vertices; i++) {
        node *temp = g->adj_lists[i];
        while(temp) {
            csr->col_indices[idx++] = temp->vertex;
            temp = temp->next;
        }
    }
}

// CUDA kernel to initialize distances to INT_MAX
__global__ void set_int_max(int* arr, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) arr[idx] = INT_MAX;
}

__global__ void bfs_kernel_bitmap(
    const int *row_offsets,
    const int *col_indices,
    int *distances,
    unsigned int *frontier_bitmap,
    unsigned int *next_frontier_bitmap,
    int level,
    int num_vertices)
{
    int v = blockIdx.x * blockDim.x + threadIdx.x;
    if(v >= num_vertices) return;

    // Check if vertex is in current frontier
    if(!(frontier_bitmap[v/32] & (1U << (v%32)))) return;

    int start = row_offsets[v];
    int end = row_offsets[v+1];

    for(int i = start; i < end; i++) {
        int neighbor = col_indices[i];
        if(atomicCAS(&distances[neighbor], INT_MAX, level) == INT_MAX) {
            atomicOr(&next_frontier_bitmap[neighbor/32], 1U << (neighbor%32));
        }
    }
}

double benchmark_bfs_gpu_bitmap(csr_graph *csr, int source, int* distances) {
    // Device allocations
    int *d_row, *d_col, *d_dist;
    unsigned int *d_curr_bitmap, *d_next_bitmap;
    hipMalloc(&d_row, (csr->num_vertices+1)*sizeof(int));
    hipMalloc(&d_col, csr->num_edges*sizeof(int));
    hipMalloc(&d_dist, csr->num_vertices*sizeof(int));

    const int bitmap_len = (csr->num_vertices + 31)/32;
    const int bitmap_size = bitmap_len * sizeof(unsigned int);
    hipMalloc(&d_curr_bitmap, bitmap_size);
    hipMalloc(&d_next_bitmap, bitmap_size);

    // Copy graph data
    hipMemcpy(d_row, csr->row_offsets, (csr->num_vertices+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, csr->col_indices, csr->num_edges*sizeof(int), hipMemcpyHostToDevice);

    // Prepare initial bitmap on host
    unsigned int *host_init_bitmap = (unsigned int*)calloc(bitmap_len, sizeof(unsigned int));
    host_init_bitmap[source/32] |= 1U << (source%32);

    // warmup

    // Reset device state
    set_int_max<<<(csr->num_vertices + 255)/256, 256>>>(d_dist, csr->num_vertices);
    hipDeviceSynchronize();
    int zero = 0;
    hipMemcpy(d_dist + source, &zero, sizeof(int), hipMemcpyHostToDevice);

    hipMemset(d_curr_bitmap, 0, bitmap_size);
    hipMemcpy(d_curr_bitmap, host_init_bitmap, bitmap_size, hipMemcpyHostToDevice);

    timer time_one_iteration;
    timer_start(&time_one_iteration);
    int level = 1;
    bool active = true;
    unsigned int *h_next_bitmap = (unsigned int*)malloc(bitmap_size);
    while(active) {
        hipMemset(d_next_bitmap, 0, bitmap_size);

        dim3 block(256);
        dim3 grid((csr->num_vertices + block.x - 1) / block.x);

        bfs_kernel_bitmap<<<grid, block>>>(
            d_row, d_col, d_dist,
            d_curr_bitmap, d_next_bitmap,
            level, csr->num_vertices
        );
        hipDeviceSynchronize();

        // Check frontier activity
        hipMemcpy(h_next_bitmap, d_next_bitmap, bitmap_size, hipMemcpyDeviceToHost);

        active = false;
        for(int i = 0; i < bitmap_len; i++) {
            if(h_next_bitmap[i] != 0) {
                active = true;
                break;
            }
        }

        // Swap bitmaps
        unsigned int *temp = d_curr_bitmap;
        d_curr_bitmap = d_next_bitmap;
        d_next_bitmap = temp;
        level++;
    }
    double estimated_time = seconds_elapsed(&time_one_iteration);

    // determine # of seconds dynamically
    int num_iterations;
    if (estimated_time == 0)
        num_iterations = MAX_ITER;
    else {
        num_iterations = (int)(TIME_LIMIT / estimated_time);
        if (num_iterations < MIN_ITER) num_iterations = MIN_ITER;
        if (num_iterations > MAX_ITER) num_iterations = MAX_ITER;
    }
    printf("\tPerforming %d iterations\n", num_iterations);

    // time several BFS iterations
    timer t;
    timer_start(&t);
    for(int j = 0; j < num_iterations; j++) {
        // Reset state for each iteration
        set_int_max<<<(csr->num_vertices + 255)/256, 256>>>(d_dist, csr->num_vertices);
        hipDeviceSynchronize();
        int zero = 0;
        hipMemcpy(d_dist + source, &zero, sizeof(int), hipMemcpyHostToDevice);

        hipMemset(d_curr_bitmap, 0, bitmap_size);
        hipMemcpy(d_curr_bitmap, host_init_bitmap, bitmap_size, hipMemcpyHostToDevice);

        level = 1;
        active = true;
        while(active) {
            hipMemset(d_next_bitmap, 0, bitmap_size);

            dim3 block(256);
            dim3 grid((csr->num_vertices + block.x - 1) / block.x);

            bfs_kernel_bitmap<<<grid, block>>>(
                d_row, d_col, d_dist,
                d_curr_bitmap, d_next_bitmap,
                level, csr->num_vertices
            );
            hipDeviceSynchronize();

            // Check frontier activity
            hipMemcpy(h_next_bitmap, d_next_bitmap, bitmap_size, hipMemcpyDeviceToHost);

            active = false;
            for(int i = 0; i < bitmap_len; i++) {
                if(h_next_bitmap[i] != 0) {
                    active = true;
                    break;
                }
            }

            // Swap bitmaps
            unsigned int *temp = d_curr_bitmap;
            d_curr_bitmap = d_next_bitmap;
            d_next_bitmap = temp;
            level++;
        }
    }
    double msec_per_iteration = milliseconds_elapsed(&t) / (double) num_iterations;
    printf("\tbenchmarking GPU-Bitmap-Breadth-First-Search: %8.4f ms \n", msec_per_iteration);

    // Copy the result back to host.
    hipMemcpy(distances, d_dist, csr->num_vertices*sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    free(host_init_bitmap);
    free(h_next_bitmap);
    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_dist);
    hipFree(d_curr_bitmap);
    hipFree(d_next_bitmap);

    return msec_per_iteration;
}

int main(int argc, char** argv) {
    if (get_arg(argc, argv, "help") != NULL){
        usage(argc, argv);
        return 0;
    }

    char * mm_filename = NULL;
    if (argc == 1) {
        printf("Give a MatrixMarket file.\n");
        return -1;
    } else 
        mm_filename = argv[1];

    graph g;
    printf("Filename: %s\n", mm_filename);
    read_graph_matrix(&g, mm_filename);

#ifdef TESTING
    //print in adjacency list format
        printf("Writing matrix in adjacency list format to test_adj_list ...");
        FILE *fp = fopen("test_adj_list", "w");
        fprintf(fp, "%d\n", g.num_vertices);
        for (int v = 0; v < g.num_vertices; v++) {
            node* temp = g.adj_lists[v];
            fprintf(fp, "Vertex %d:", v);
            while (temp) {
                fprintf(fp, " -> %d", temp->vertex);
                temp = temp->next;
            }
            fprintf(fp, "\n");
        }
        fclose(fp);
        printf("... done!\n");
#endif 
    
    csr_graph csr;
    convert_to_csr(&g, &csr);
    delete_graph(&g);

    int* distances = (int *)malloc(csr.num_vertices * sizeof(int));
    int source = 0;
    benchmark_bfs_gpu_bitmap(&csr, source, distances);

/* Test correctnesss */
#ifdef TESTING
    printf("Writing distance values from source: %d ...", source);
    fp = fopen("test_dist_cuda", "w");
    for (int i=0; i<csr.num_vertices; i++)
    {
      fprintf(fp, "Node %d : %d\n", i, distances[i]);
    }
    fclose(fp);
    printf("... done!\n");
#endif 

    delete_csr_graph(&csr);    
    return 0;
}
